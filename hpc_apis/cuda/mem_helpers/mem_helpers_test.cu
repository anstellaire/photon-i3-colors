#include "hip/hip_runtime.h"
#include "mem_helpers.cuh"

#include <cassert>
#include <cstdint>
#include <vector>

template<typename T, typename F, typename K, typename C>
void test_device_memory(std::size_t num, F filler, K kernel, C checker) {
    // allocate host memory
    auto host_mem = std::vector<T>(num);

    // allocate device memory
    auto dev_mem = device_memory<T>(num);

    // launch filler
    filler(host_mem.data(), host_mem.size());

    // move from host to device
    dev_mem.from_host(host_mem.data());

    // launch kernel
    kernel(dev_mem.data(), dev_mem.size());

    // move from device to host
    dev_mem.to_host(host_mem.data());

    // launch checker
    checker(host_mem.data(), host_mem.size());
}

template<typename T, typename F, typename K, typename C>
void test_shared_memory(std::size_t num, F filler, K kernel, C checker) {
    // allocate shared memory
    auto shared_mem = shared_memory<T>(num);

    // launch filler
    filler(shared_mem.data(), shared_mem.size());

    // [optional] prefetch from host to device
    int dev_id;
    hipGetDevice(&dev_id);
    shared_mem.advise(hipMemAdviseSetReadMostly, dev_id);
    shared_mem.prefetch_device(dev_id);

    // launch kernel
    kernel(shared_mem.data(), shared_mem.size());

    // wait kernel completion
    hipDeviceSynchronize();

    // [optional] prefetch from device to host
    shared_mem.prefetch_host();

    // launch checker
    checker(shared_mem.data(), shared_mem.size());
}

template<typename T, typename F, typename K, typename C>
void test_pinned_memory(std::size_t num, F filler, K kernel, C checker) {
    // allocate host memory
    auto pinned_mem = pinned_memory<T>(num);

    // allocate device memory
    auto dev_mem = device_memory<T>(num);

    // launch filler
    filler(pinned_mem.data(), pinned_mem.size());

    // move from host to device
    dev_mem.from_host(pinned_mem.data());

    // launch kernel
    kernel(dev_mem.data(), dev_mem.size());

    // move from device to host
    dev_mem.to_host(pinned_mem.data());

    // launch checker
    checker(pinned_mem.data(), pinned_mem.size());
}

template<typename T>
__global__ void test_kernel(T* vec, std::size_t num) {
    std::size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < num)
        vec[i] = i;
}

int main() {
    using test_type = int;

    auto filler = [](test_type* host_mem, std::size_t num) {
        for (std::size_t i = 0; i < num; ++i)
            host_mem[i] = 0;
    };

    auto checker = [](test_type* host_mem, std::size_t num) {
        for (std::size_t i = 0; i < num; ++i)
            assert(host_mem[i] == i);
    };

    auto kernel = [](test_type* dev_mem, std::size_t num) {
        std::size_t const block_size = 32 * 32;
        std::size_t const grid_size = (num + block_size - 1) / block_size;
        test_kernel<<<grid_size, block_size>>>(dev_mem, num);
    };

    test_device_memory<test_type>(1 << 16, filler, kernel, checker);
    test_shared_memory<test_type>(1 << 16, filler, kernel, checker);
    test_pinned_memory<test_type>(1 << 16, filler, kernel, checker);
}
