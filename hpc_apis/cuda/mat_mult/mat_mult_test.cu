#include "mat_mult.cuh"

#include <vector>
#include <memory>

template<typename T>
std::unique_ptr<T, void(*)(T*)> create_dev_mem(std::size_t num) {
    T* ptr = NULL;
    hipMalloc(&ptr, num*sizeof(T));

    return std::unique_ptr<T, void(*)(T*)>{ptr, [](T* ptr){
        hipFree(ptr);
    }};
}

template<typename Type>
void test_for(std::size_t const a_height, std::size_t const a_width, std::size_t const b_width) {

    // allocate host memory
    std::vector<Type> host_a(a_height*a_width);
    std::vector<Type> host_b(a_width*b_width);
    std::vector<Type> host_c(a_height*b_width);

    // fill host memory
    for(Type& val : host_a) val = std::rand() % 100;
    for(Type& val : host_b) val = std::rand() % 100;

    // allocate device memory
    auto dev_a = create_dev_mem<Type>(a_height*a_width);
    auto dev_b = create_dev_mem<Type>(a_width*b_width);
    auto dev_c = create_dev_mem<Type>(a_height*b_width);

    // move from host to device
    hipMemcpy(dev_a.get(), host_a.data(), host_a.size()*sizeof(Type), hipMemcpyHostToDevice);
    hipMemcpy(dev_b.get(), host_b.data(), host_b.size()*sizeof(Type), hipMemcpyHostToDevice);

    // launch kernel
    mat_mult(dev_a.get(), dev_b.get(), dev_c.get(), a_height, a_width, b_width);

    // move from device to host
    hipMemcpy(host_c.data(), dev_c.get(), host_c.size()*sizeof(Type), hipMemcpyDeviceToHost);

    // check result
    verify_mat_mult(host_a.data(), host_b.data(), host_c.data(), a_height, a_width, b_width);
}

int main() {
    test_for<int>(/*a_height*/1024, /*a_width*/2048, /*b_width*/512);
}
